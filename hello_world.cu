
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void hello_world() {
    // printf("Hello World from Cuda\n");
    // printf("threadIdx.x: %d, threadIdx.y: %d, threadIdx.z\n", threadIdx.x, threadIdx.y, threadIdx.z);
    printf("threadIdx.x: %d, threadIdx.y: %d, threadIdx.z: %d <-> blockIdx.x: %d, blockIdx.y: %d, blockIdx.z: %d \n", threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y, blockIdx.z);
    
}

int main() {
    dim3 grid(4, 1, 1);
    dim3 block(2, 1, 1);


    hello_world<<<block, grid>>>();


    hipDeviceSynchronize();

    hipDeviceReset();

    return 0;
}