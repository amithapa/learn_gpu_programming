
#include <hip/hip_runtime.h>
#include<stdio.h>
#define ARRAY_SIZE 16

__global__ void print_index_and_data(int * data) {
    int tid = threadIdx.x;
    int block_offeset = blockIdx.x * blockDim.x;
    int row_offset = blockDim.x * gridDim.x * blockIdx.y;
    int gid = tid + block_offeset + row_offset;
    // printf("threadIdx.x: %d, offeset: %d, gid: %d, blockIdx.x: %d, blockDim.x: %d, gridDim.x: %d, data: %d\n", tid, gid, offeset, blockIdx.x, blockDim.x, gridDim.x, data[gid]);
    //printf("threadIdx.x: %d, gid: %d, blockIdx.x: %d, blockIdx.y: %d, blockDim.x: %d, blockDim.y: %d, gridDim.x: %d, data: %d\n", tid, gid, blockIdx.x, blockIdx.y, blockDim.x, blockDim.y, gridDim.x, data[gid]);
    printf("%d\t\t%d\t\t%d\t\t%d\t\t%d\t\t%d\t\t%d\t%d\n", tid, gid, blockIdx.x, blockIdx.y, blockDim.x, blockDim.y, gridDim.x, data[gid]);
}

int main() {
    // int nx = 16;
    // int ny = 16;

    int array_size_bytes = sizeof(int) * ARRAY_SIZE;
    // int h_data[ARRAY_SIZE] = {23, 9, 7, 14, 27, 4, 3, 11, 10, 13, 61, 42, 50, 67, 83, 22};
    int h_data[ARRAY_SIZE]; 
    for(int i=0; i<ARRAY_SIZE; i++) {
        h_data[i] =i;
        printf("%d ", h_data[i]);
    }

    printf("\n\n");

    dim3 block(4);
    dim3 grid(2, 2);

    int * d_data;
    hipMalloc((void**)&d_data, array_size_bytes);

    hipMemcpy(d_data, h_data, array_size_bytes, hipMemcpyHostToDevice);
    printf("threadIdx.x\tgid\tblockIdx.x\tblockIdx.y\tblockDim.x\tblockDim.y\tgridDim.x\tdata\n");
    print_index_and_data<<<grid, block>>>(d_data);
    // print_index_and_data<<<4, 4>>>(d_data);
    
    hipDeviceSynchronize();
    hipDeviceReset();
    

    return 0;
}