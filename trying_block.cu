
#include <hip/hip_runtime.h>
#include<stdio.h>

#define NUM_BLOCKS 15
#define BLOCK_WIDTH 1

__global__ void hello() {
    printf("Hello world! I am a thread block %d\n", blockIdx.x);
}

int main(int argc, char **argv) {
    // Launch the kernal
    hello<<<NUM_BLOCKS, BLOCK_WIDTH>>>();

    // force the printf()s to flush
    hipDeviceSynchronize();

    printf("that is all\n");

    return 0;
}