
#include <hip/hip_runtime.h>
#include<stdio.h>
#define ARRAY_SIZE 16

__global__ void print_index_and_data(int * data) {
    int tid = threadIdx.y  * blockDim.x + threadIdx.x;

    int number_of_threads_in_block = blockDim.x * blockDim.y;

    int block_offeset = blockIdx.x * number_of_threads_in_block;

    int number_of_threads_in_row = number_of_threads_in_block * gridDim.x; 
    
    int row_offset = number_of_threads_in_row * blockIdx.y;
    int gid = tid + block_offeset + row_offset;
    
    printf("tid: %d, block_offset: (%d * %d = %d), row_offset  (%d * %d = %d) = %d\n", tid, number_of_threads_in_block, blockIdx.x, block_offeset, number_of_threads_in_row, blockIdx.y, row_offset, gid);

    printf("threadIdx.x: %d, threadIdx.y: %d, blockDim.x: %d, tid: %d, gid: %d, blockIdx.x: %d,, blockIdx.y: %d,, blockDim.x: %d,, blockDim.y: %d, gridDim.x: %d, gridDim.y: %d, data: %d\n", 
    threadIdx.x, threadIdx.y, blockDim.x, tid, gid, blockIdx.x, blockIdx.y, blockDim.x, blockDim.y, gridDim.x, gridDim.y, data[gid]);
}

int main() {

    int array_size_bytes = sizeof(int) * ARRAY_SIZE;
    int h_data[ARRAY_SIZE]; 
    for(int i=0; i<ARRAY_SIZE; i++) {
        h_data[i] =i;
        printf("%d ", h_data[i]);
    }

    printf("\n\n");

    dim3 block(2, 2);
    dim3 grid(2, 2);

    int * d_data;
    hipMalloc((void**)&d_data, array_size_bytes);

    hipMemcpy(d_data, h_data, array_size_bytes, hipMemcpyHostToDevice);
    print_index_and_data<<<grid, block>>>(d_data);

    
    hipDeviceSynchronize();
    hipDeviceReset();
    

    return 0;
}