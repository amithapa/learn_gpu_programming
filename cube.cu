
#include <hip/hip_runtime.h>
# include <stdio.h>

__global__ void cube(float *d_in, float *d_out) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    float f = d_in[idx];
    d_out[idx] = f * f *f;
}

int main() {
    const int ARRAY_SIZE = 4000;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

    float h_in[ARRAY_SIZE];
    for(int i=0; i < ARRAY_SIZE; i++) {
        h_in[i] = float(i);
    }
    float h_out[ARRAY_SIZE];

    // declare memory pointers
    float * d_in;
    float * d_out;

    // allocating memory for GPU variables
    hipMalloc((void **) &d_in, ARRAY_BYTES);
    hipMalloc((void **) &d_out, ARRAY_BYTES);

    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);
    
    cube<<<4, 1000>>>(d_in, d_out);

    hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

    for(int i=0; i < ARRAY_SIZE; i++) {
        printf("%f", h_out[i]);
        printf(((i%4) != 3) ? "\t" : "\n");
    }

    hipFree(d_in);
    hipFree(d_out);

    return 0;
}