
#include <hip/hip_runtime.h>
#include<stdio.h>


__global__ void print_indexs() {
    printf("threadIdx.x: %d, threadIdx.y: %d, threadIdx.z: %d, blockIdx.x: %d, blockIdx.y: %d, blockIdx.z: %d, blockDim.x: %d, blockDim.y: %d, blockDim.z: %d, gridDim.x: %d, gridDim.y: %d, gridDim.z: %d\n",
    threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z);
}

int main() {
    int nx = 16;
    int ny = 16;
    int nz = 16;

    dim3 block(8,8,8);
    dim3 grid(nx/block.x, ny/block.y, nz/block.z);
    print_indexs<<<grid, block>>>();
    hipDeviceSynchronize();
    hipDeviceReset();

    return 0;
}