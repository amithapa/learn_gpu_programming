
#include <hip/hip_runtime.h>
# include<stdio.h>

__global__ void print_thread_ids() {
    printf("threadIdx.x: %d, threadIdx.y: %d, threadIdx.z: %d <-> blockIdx.x: %d, blockIdx.y: %d, blockIdx.z: %d <-> blockDim.x: %d, blockDim.y: %d, blockDim.z: %d <-> gridDim.x: %d, gridDim.y: %d, gridDim.z: %d\n", 
    threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z);
    // printf("threadIdx.x: %d, threadIdx.y: %d, threadIdx.z: %d\n", threadIdx.x, threadIdx.y, threadIdx.z);
}

int main() {

    int nx, ny;
    nx = 16;
    ny = 16;

    dim3 block(8,8);
    dim3 grid(nx/block.x, ny/block.y);

    print_thread_ids<<<grid, block>>>();

    hipDeviceSynchronize();
    hipDeviceReset();
    return 0;
}