
#include <hip/hip_runtime.h>
#include<stdio.h>
#define ARRAY_SIZE 16

__global__ void unique_idx_calc_threadIdx(int * input) {
    int tid = threadIdx.x+blockIdx.x*blockDim.x;
    printf("threadIdx.x: %d, blockIdx.x: %d, blockDim.x: %d, value: %d\n", tid, blockIdx.x, blockDim.x, input[tid]);
}

__global__ void unique_gid_calc(int * input) {
    int tid = threadIdx.x;
    int offset = blockDim.x * blockIdx.x;
    int gid = tid + offset;
    printf("threadIdx.x: %d, blockIdx.x: %d, gid.x: %d, value: %d\n", tid, blockIdx.x, gid, input[gid]);
}
int main() {

    int ARRAY_BYTES = ARRAY_SIZE * sizeof(int);

    int h_data[] = {23, 9, 7, 14, 27, 4, 3, 11, 10, 13, 61, 42, 50, 67, 83, 22};
    for(int i=0; i<ARRAY_SIZE; i++) {
        printf("%d ", h_data[i]);
    }
    printf("\n \n");
    int * d_data;
    hipMalloc((void**)&d_data, ARRAY_BYTES);
    hipMemcpy(d_data, h_data, ARRAY_BYTES, hipMemcpyHostToDevice);

    dim3 grid(4);
    dim3 block(4);

    // unique_idx_calc_threadIdx<<<grid,block>>>(d_data);
    unique_gid_calc<<<grid,block>>>(d_data);
    hipDeviceSynchronize();
    hipDeviceReset();

    return 0;
}